#include "hip/hip_runtime.h"

#include<thrust/scan.h>
#include<thrust/sort.h>
#include <thrust/unique.h>
#include<thrust/copy.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<vector>
#include<set>
#include<algorithm>
#include<numeric>
#include<set>
#include<iostream>
#include<stdio.h>
#include<fstream>
#include <sys/time.h>
#include<hip/hip_runtime.h>
using namespace std;


//structure which stores source vertex,target vertex and weight....

struct edgeingraph{
    int u;
    int v;
    int w;
};
bool comp(edgeingraph o1, edgeingraph o2)
    {
  
        if(o1.u==o2.u){
          return (o1.w<o2.w);
        }
        return (o1.u<o2.u);
    }
//set is just for finding number of vertices....
set<int> SET;
int verticescount_n=0;
int edgescount_m=0;
vector<struct edgeingraph> edgeStream;


//this function will read the input from file
//one edge is added twice
//example if 0 1 2 is taken 
//then 1 0 2 is also added
//making it easier to generate csr representation

void readEdgeStream(){
    ifstream fin;
    fin.open("edgesinput.txt"); //can give your input file name here
    struct edgeingraph edge;
    struct edgeingraph edge2;
        fin>>edge.u;
        SET.insert(edge.u);
        fin>>edge.v;
        SET.insert(edge.v);
        fin>>edge.w;
        edgescount_m++;
        edge2.u=edge.v;
        edge2.v=edge.u;
        edge2.w=edge.w;
        if(edge.u!=edge.v){
        edgeStream.push_back(edge);
        edgeStream.push_back(edge2);
        }
    while (fin) {
      struct edgeingraph edge1;
      struct edgeingraph edge3;
        fin>>edge1.u;
        if( fin.eof() ) break;
        SET.insert(edge1.u);
        fin>>edge1.v;
        SET.insert(edge1.v);
        fin>>edge1.w;
        edgescount_m++;
        edge3.u=edge1.v;
        edge3.v=edge1.u;
        edge3.w=edge1.w;
        if(edge1.u!=edge1.v){
        edgeStream.push_back(edge1);
        edgeStream.push_back(edge3);
        }
    }
    fin.close();
}

//will create and sorts edgestreamarray
//from edgeStream vector

void sortEdgeStream(struct edgeingraph edgestreamarray[]){
    int *keys=new int[edgescount_m*2];
    for(int i=0;i<2*edgescount_m;i++){
      keys[i]=static_cast<struct edgeingraph>(edgeStream[i]).u;
      edgestreamarray[i].u=edgeStream[i].u;
      edgestreamarray[i].v=edgeStream[i].v;
      edgestreamarray[i].w=edgeStream[i].w;
    }
    thrust::sort_by_key(keys,keys+(2*edgescount_m),edgestreamarray);
    delete[] keys;
}



void edgeStreamToCSR(int edgesarray[],int verticesarray[], int weightsarray[],struct edgeingraph edgestreamarray[]){
  for(int i=0;i<2*edgescount_m;i++){
        edgesarray[i]=edgestreamarray[i].v;
  }

  for(int i=0;i<2*edgescount_m;i++){
        weightsarray[i]=edgestreamarray[i].w;
  }
  
  int it=0;
  for(int i=0;i<2*edgescount_m;i++){
   if(edgestreamarray[i].u==it){
     verticesarray[it]=i;
     it++;
   }     
  }
  verticesarray[verticescount_n]=2*edgescount_m;

}



__global__ void createSusscessorArray(int *su,int *w,int *ed,int *v,int n,int m)
{
    int tid=(blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<n&& tid>=0){
    int minindex=v[tid];
    for(int i=v[tid]+1;i<v[tid+1];i++){
        if(w[i]<w[minindex]){
          minindex=i;
        }
    }
    su[tid]=ed[minindex];
    }
}

__global__ void createSuperVertices(int *su,int n)
{
    int tid=(blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<n&& tid>=0){
    while(su[tid]!=su[su[tid]]){
      su[tid]=su[su[tid]];
    }
    }
}
__global__ void createnewedgestream(int *s,edgeingraph *edsa, int n,int m,int *ts,int *ind)
{
    int tid=(blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid<2*m && tid>=0){
      (edsa[tid]).u=ind[s[(edsa[tid]).u]];
      (edsa[tid]).v=ind[s[(edsa[tid]).v]];
    }
}

int main(){

float totaltime=0;
  cout<<"EDGES OF MINIMUM SPANNING TREE: "<<endl;
  int cost=0;
  int count=0;
  readEdgeStream();


  //cleans multigraph as simple graph by deleting larger weights and keeping small weight between two vertices.

  for (auto i = edgeStream.begin(); i != edgeStream.end(); ++i) {
    for(auto j=i+1;j != edgeStream.end();++j){
      if((*j).u==(*i).u && (*j).v == (*i).v){
        if((*j).w>=(*i).w){
          edgeStream.erase(j);
          j--;
        }
        else{
             edgeStream.erase(i);
             i--;break;
        }
      }
    }  
}
edgescount_m=edgeStream.size()/2;

  int f=SET.size();
  int flag[f];
  for(int i=0;i<f;i++){
    flag[i]=i;
  }
  vector<struct edgeingraph> backup;
  backup=edgeStream;



while(count<f-1){
  verticescount_n=SET.size();
  struct edgeingraph edgestreamarray[2*edgescount_m];
  sortEdgeStream(edgestreamarray);
  sort(edgestreamarray,edgestreamarray+2*edgescount_m,comp);
  int *edgesarray=new int[2*edgescount_m];
  int *weightsarray=new int[2*edgescount_m];
  int *verticesarray=new int[verticescount_n+1];
  edgeStreamToCSR(edgesarray,verticesarray,weightsarray,edgestreamarray);

  int *successorarray=new int[verticescount_n];
    int *v;
    hipMalloc((void **)&v, (verticescount_n+1)*sizeof(int));
    int *ed;
    hipMalloc((void **)&ed, (edgescount_m*2)*sizeof(int));
    int *w;
    hipMalloc((void **)&w, (edgescount_m*2)*sizeof(int));
    int *su;
    hipMalloc((void **)&su, (verticescount_n+1)*sizeof(int));
    

  hipMemcpy(v, verticesarray, (verticescount_n+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ed, edgesarray, (edgescount_m*2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(w, weightsarray, (edgescount_m*2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(su, successorarray, (verticescount_n)*sizeof(int), hipMemcpyHostToDevice);
  int th=(verticescount_n+1023)/1024;
  float time=0;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  createSusscessorArray<<<th, 1024>>>(su,w,ed,v,verticescount_n,edgescount_m);
hipEventRecord(stop,0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&time,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  totaltime+=time;
  cout<<endl<<endl<<time<<endl<<endl;
  hipMemcpy(successorarray,su,(verticescount_n)*sizeof(int), hipMemcpyDeviceToHost);
hipFree(v);
hipFree(ed);
hipFree(w);


 for(int i=0;i<verticescount_n;i++){
   if(successorarray[successorarray[i]]==i){
     successorarray[successorarray[i]]=successorarray[i];
   }
 }
 for(int i=0;i<verticescount_n;i++){
   if(successorarray[i]!=i){
     int weight=0;
     for(int j=verticesarray[i];j<verticesarray[i+1];j++){
       if(edgesarray[j]==successorarray[i]){
         weight=weightsarray[j];
       }
     }
    for(int k=0;k<backup.size();k++){
        if(backup[k].w==weight){
          if((flag[backup[k].u]==i && flag[backup[k].v]==successorarray[i]) || (flag[backup[k].v]==i && flag[backup[k].u]==successorarray[i])){
            cout<<backup[k].u<<" "<<backup[k].v<<" "<<weight<<endl;
            break;
          }
        }
    }
    cost+=weight;
    count++;
   }
 }

  hipMemcpy(su, successorarray, (verticescount_n)*sizeof(int), hipMemcpyHostToDevice);
  time=0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  createSuperVertices<<<th, 1024>>>(su,verticescount_n);
  hipEventRecord(stop,0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&time,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  totaltime+=time;
  cout<<endl<<endl<<time<<endl<<endl;
  hipMemcpy(successorarray,su,(verticescount_n)*sizeof(int), hipMemcpyDeviceToHost);
hipFree(su);
//relabeling

int *tempsuc=new int[verticescount_n];
thrust::copy(successorarray,successorarray+verticescount_n,tempsuc);
thrust::sort(tempsuc,tempsuc+verticescount_n);
thrust::unique(tempsuc,tempsuc+verticescount_n);

int index[verticescount_n]={-1};
fill(index,index+verticescount_n,-1);
for(int i=0;i<verticescount_n;i++){
  if(index[tempsuc[i]]==-1){
  index[tempsuc[i]]=i;
  }
}
for(int i=0;i<f;i++){

  int temp=flag[i];
   
  flag[i]=index[successorarray[temp]];

}
int *ind;
hipMalloc((void **)&ind, verticescount_n*sizeof(int));
int *ts;
hipMalloc((void **)&ts, sizeof(tempsuc));
int *s;
hipMalloc((void **)&s, (verticescount_n)*sizeof(int));
struct edgeingraph *edsa;
hipMalloc((void **)&edsa, (2*edgescount_m)*sizeof(edgeingraph));
th=((2*edgescount_m)+1023)/1024;
hipMemcpy(ind,index,verticescount_n*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(ts,tempsuc,sizeof(tempsuc),hipMemcpyHostToDevice);
hipMemcpy(edsa, edgestreamarray, (edgescount_m*2)*sizeof(edgeingraph), hipMemcpyHostToDevice);
hipMemcpy(s, successorarray, (verticescount_n)*sizeof(int), hipMemcpyHostToDevice);
 time=0; 
 /*hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);*/
createnewedgestream<<<th, 1024>>>(s,edsa,verticescount_n,edgescount_m,ts,ind);
/*hipEventRecord(stop,0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&time,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  totaltime+=time;*/
  //cout<<endl<<endl<<time<<endl<<endl;
hipMemcpy(edgestreamarray, edsa,(edgescount_m*2)*sizeof(edgeingraph), hipMemcpyDeviceToHost);
hipFree(s);
hipFree(edsa);
hipFree(ts);
hipFree(ts);
edgeStream.clear();
SET.clear();
edgeStream.insert(edgeStream.begin(),edgestreamarray,edgestreamarray+edgescount_m*2);
 for (auto i = edgeStream.begin(); i != edgeStream.end(); ++i) {
     SET.insert((*i).u);
     SET.insert((*i).v);
        if ((*i).u == (*i).v) {
            edgeStream.erase(i);
            i--;
        }
}
 for (auto i = edgeStream.begin(); i != edgeStream.end(); ++i) {
    for(auto j=i+1;j != edgeStream.end();++j){
      if((*j).u==(*i).u && (*j).v == (*i).v){
        if((*j).w>=(*i).w){
          edgeStream.erase(j);
          j--;
        }
        else{
             edgeStream.erase(i);
             i--;break;
        }
      }
    }  
}
edgescount_m=edgeStream.size()/2;
}
cout<<endl<<"TOTAL COST = "<<cost<<endl;
cout<<totaltime<<endl;
}
